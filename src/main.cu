

#include <hip/hip_runtime.h>
#include <stdio.h>
#define CHECK(e) { int res = (e); if (res) printf("CUDA ERROR %d\n", res); }
#define THRESH 10000

#define N 480

#define WIDTH 640
#define HEIGHT 480

struct Image {
    int width;
    int height;
    unsigned char *img;
    unsigned char *dev_img;
};

unsigned char getPixel(Image *image, int x, int y)
{
    int id = x * image->width + y;
    return image->img[id];
}

void setPixel(Image *image, int x, int y, unsigned char value)
{
    int id = x * image->width + y;
    image->img[id] = value;
}


    __device__
unsigned char getPixel2(unsigned char *image, int x, int y, int width)
{
    int id = y * WIDTH + x;
    return image[id];
}

    __device__
void setPixel2(unsigned char *image, int x, int y, unsigned char value, int width)
{
    int id = y * WIDTH + x;
    image[id] = value;
}

    __global__
void filter(unsigned char *grayScale, unsigned char *filtered)
{
    /*int tid = (blockIdx.x * gridDim.y * blockDim.x * blockDim.y) + (blockIdx.y * blockDim.x * blockDim.y) + (threadIdx.x * blockDim.y) + threadIdx.y;*/

    /*int i = tid / N;*/
    /*int j = tid % N;*/

    int i = blockIdx.x;
    int j = blockIdx.y;

    if (i == 0 || i == WIDTH - 1 || j == 0 || j == HEIGHT - 1)
    {
        return;
    }

    int gradX = getPixel2(grayScale, i-1, j+1, N) - getPixel2(grayScale, i-1, j-1, N) + 2*getPixel2(grayScale, i, j+1, N) - 2*getPixel2(grayScale, i, j-1, N) + getPixel2(grayScale, i+1, j+1, N) - getPixel2(grayScale, i+1, j-1, N);

    int gradY = getPixel2(grayScale, i-1, j-1, N) + 2*getPixel2(grayScale, i-1, j, N) + getPixel2(grayScale, i-1, j+1, N) - getPixel2(grayScale, i+1, j-1, N) - 2*getPixel2(grayScale, i+1, j, N) - getPixel2(grayScale, i+1, j+1, N);

    int magnitude = gradX*gradX + gradY*gradY;

    if (magnitude  > 10000)
    {
        setPixel2(filtered, i, j, 255, N);
    }
    else
    {
        setPixel2(filtered, i, j, 0, N);
    }
}



int main(int argc, char **argv)
{
    Image source;

    if (argc != 3)
    {
        printf("Usage: exec filename filename\n");
        exit(1);
    }
    char *fname = argv[1];
    char *fname2 = argv[2];
    FILE *src;

    if (!(src = fopen(fname, "rb")))
    {
        printf("Couldn't open file %s for reading.\n", fname);
        exit(1);
    }

    char p,s;
    fscanf(src, "%c%c\n", &p, &s);
    if (p != 'P' || s != '6')
    {
        printf("Not a valid PPM file (%c %c)\n", p, s);
        exit(1);
    }

    fscanf(src, "%d %d\n", &source.width, &source.height);
    int ignored;
    fscanf(src, "%d\n", &ignored);

    int pixels = source.width * source.height;
    source.img = (unsigned char *)malloc(pixels*3);
    if (fread(source.img, sizeof(unsigned char), pixels*3, src) != pixels*3)
    {
        printf("Error reading file.\n");
        exit(1);
    }
    fclose(src);

    Image grayScale;
    grayScale.width = source.width;
    grayScale.height = source.height;
    grayScale.img = (unsigned char *)malloc(pixels);
    for (int i = 0; i < pixels; i++)
    {
        unsigned int r = source.img[i*3];
        unsigned int g = source.img[i*3 + 1];
        unsigned int b = source.img[i*3 + 2];
        grayScale.img[i] = 0.2989*r + 0.5870*g + 0.1140*b;
    }

    Image filtered;
    filtered.width = source.width;
    filtered.height = source.height;
    filtered.img = (unsigned char *)malloc(pixels);

    for (int i = 0; i < pixels; i++)
    {
        filtered.img[i] = 0;
    }

    unsigned char *devGrayScale;
    unsigned char *devFiltered;


    // Initialize Cuda Memory
    CHECK(hipMalloc(&devGrayScale, WIDTH * HEIGHT * sizeof(unsigned char)));
    CHECK(hipMalloc(&devFiltered, WIDTH * HEIGHT * sizeof(unsigned char)));

    /*// Copy Cuda Memory*/
    CHECK(hipMemcpy(devGrayScale, grayScale.img, WIDTH * HEIGHT * sizeof(unsigned char), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(devFiltered, filtered.img, WIDTH * HEIGHT * sizeof(unsigned char), hipMemcpyHostToDevice));

    /*// Run the kernel*/
    dim3 dimBlock(WIDTH, HEIGHT);
    dim3 dimGrid(1);
    filter<<<dimBlock, dimGrid>>>(devGrayScale, devFiltered);

    /*// Return the Cuda Memory*/
    CHECK(hipMemcpy(filtered.img, devFiltered, WIDTH * HEIGHT * sizeof(unsigned char), hipMemcpyDeviceToHost));


    FILE *out;
    if (!(out = fopen(fname2, "wb")))
    {
        printf("Couldn't open file for output.\n");
        exit(1);
    }
    fprintf(out, "P5\n%d %d\n255\n", filtered.width, filtered.height);
    if (fwrite(filtered.img, sizeof(unsigned char), pixels, out) != pixels)
    {
        printf("Error writing file.\n");
        exit(1);
    }
    fclose(out);

    free(grayScale.img);
    free(source.img);
    free(filtered.img);

    hipFree(devGrayScale);
    hipFree(devFiltered);

    exit(0);
}
